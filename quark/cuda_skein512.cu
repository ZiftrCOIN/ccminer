#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <stdint.h>

#include "cuda_helper.h"

#define SPH_C64(x)    ((uint64_t)(x ## ULL))

// aus cpu-miner.c
extern "C" extern int device_map[8];
extern int compute_version[8];
// aus heavy.cu
extern hipError_t MyStreamSynchronize(hipStream_t stream, int situation, int thr_id);


#define SHL(x, n)			((x) << (n))
#define SHR(x, n)			((x) >> (n))

// Zum testen Hostcode...
/*	Hier erstmal die Tabelle mit den Konstanten f�r die Mix-Funktion. Kann sp�ter vll.
	mal direkt in den Code eingesetzt werden
*/

/*
 * M9_ ## s ## _ ## i  evaluates to s+i mod 9 (0 <= s <= 18, 0 <= i <= 7).
 */

#define M9_0_0    0
#define M9_0_1    1
#define M9_0_2    2
#define M9_0_3    3
#define M9_0_4    4
#define M9_0_5    5
#define M9_0_6    6
#define M9_0_7    7

#define M9_1_0    1
#define M9_1_1    2
#define M9_1_2    3
#define M9_1_3    4
#define M9_1_4    5
#define M9_1_5    6
#define M9_1_6    7
#define M9_1_7    8

#define M9_2_0    2
#define M9_2_1    3
#define M9_2_2    4
#define M9_2_3    5
#define M9_2_4    6
#define M9_2_5    7
#define M9_2_6    8
#define M9_2_7    0

#define M9_3_0    3
#define M9_3_1    4
#define M9_3_2    5
#define M9_3_3    6
#define M9_3_4    7
#define M9_3_5    8
#define M9_3_6    0
#define M9_3_7    1

#define M9_4_0    4
#define M9_4_1    5
#define M9_4_2    6
#define M9_4_3    7
#define M9_4_4    8
#define M9_4_5    0
#define M9_4_6    1
#define M9_4_7    2

#define M9_5_0    5
#define M9_5_1    6
#define M9_5_2    7
#define M9_5_3    8
#define M9_5_4    0
#define M9_5_5    1
#define M9_5_6    2
#define M9_5_7    3

#define M9_6_0    6
#define M9_6_1    7
#define M9_6_2    8
#define M9_6_3    0
#define M9_6_4    1
#define M9_6_5    2
#define M9_6_6    3
#define M9_6_7    4

#define M9_7_0    7
#define M9_7_1    8
#define M9_7_2    0
#define M9_7_3    1
#define M9_7_4    2
#define M9_7_5    3
#define M9_7_6    4
#define M9_7_7    5

#define M9_8_0    8
#define M9_8_1    0
#define M9_8_2    1
#define M9_8_3    2
#define M9_8_4    3
#define M9_8_5    4
#define M9_8_6    5
#define M9_8_7    6

#define M9_9_0    0
#define M9_9_1    1
#define M9_9_2    2
#define M9_9_3    3
#define M9_9_4    4
#define M9_9_5    5
#define M9_9_6    6
#define M9_9_7    7

#define M9_10_0   1
#define M9_10_1   2
#define M9_10_2   3
#define M9_10_3   4
#define M9_10_4   5
#define M9_10_5   6
#define M9_10_6   7
#define M9_10_7   8

#define M9_11_0   2
#define M9_11_1   3
#define M9_11_2   4
#define M9_11_3   5
#define M9_11_4   6
#define M9_11_5   7
#define M9_11_6   8
#define M9_11_7   0

#define M9_12_0   3
#define M9_12_1   4
#define M9_12_2   5
#define M9_12_3   6
#define M9_12_4   7
#define M9_12_5   8
#define M9_12_6   0
#define M9_12_7   1

#define M9_13_0   4
#define M9_13_1   5
#define M9_13_2   6
#define M9_13_3   7
#define M9_13_4   8
#define M9_13_5   0
#define M9_13_6   1
#define M9_13_7   2

#define M9_14_0   5
#define M9_14_1   6
#define M9_14_2   7
#define M9_14_3   8
#define M9_14_4   0
#define M9_14_5   1
#define M9_14_6   2
#define M9_14_7   3

#define M9_15_0   6
#define M9_15_1   7
#define M9_15_2   8
#define M9_15_3   0
#define M9_15_4   1
#define M9_15_5   2
#define M9_15_6   3
#define M9_15_7   4

#define M9_16_0   7
#define M9_16_1   8
#define M9_16_2   0
#define M9_16_3   1
#define M9_16_4   2
#define M9_16_5   3
#define M9_16_6   4
#define M9_16_7   5

#define M9_17_0   8
#define M9_17_1   0
#define M9_17_2   1
#define M9_17_3   2
#define M9_17_4   3
#define M9_17_5   4
#define M9_17_6   5
#define M9_17_7   6

#define M9_18_0   0
#define M9_18_1   1
#define M9_18_2   2
#define M9_18_3   3
#define M9_18_4   4
#define M9_18_5   5
#define M9_18_6   6
#define M9_18_7   7

/*
 * M3_ ## s ## _ ## i  evaluates to s+i mod 3 (0 <= s <= 18, 0 <= i <= 1).
 */

#define M3_0_0    0
#define M3_0_1    1
#define M3_1_0    1
#define M3_1_1    2
#define M3_2_0    2
#define M3_2_1    0
#define M3_3_0    0
#define M3_3_1    1
#define M3_4_0    1
#define M3_4_1    2
#define M3_5_0    2
#define M3_5_1    0
#define M3_6_0    0
#define M3_6_1    1
#define M3_7_0    1
#define M3_7_1    2
#define M3_8_0    2
#define M3_8_1    0
#define M3_9_0    0
#define M3_9_1    1
#define M3_10_0   1
#define M3_10_1   2
#define M3_11_0   2
#define M3_11_1   0
#define M3_12_0   0
#define M3_12_1   1
#define M3_13_0   1
#define M3_13_1   2
#define M3_14_0   2
#define M3_14_1   0
#define M3_15_0   0
#define M3_15_1   1
#define M3_16_0   1
#define M3_16_1   2
#define M3_17_0   2
#define M3_17_1   0
#define M3_18_0   0
#define M3_18_1   1

#define XCAT(x, y)     XCAT_(x, y)
#define XCAT_(x, y)    x ## y

#define SKBI(k, s, i)   XCAT(k, XCAT(XCAT(XCAT(M9_, s), _), i))
#define SKBT(t, s, v)   XCAT(t, XCAT(XCAT(XCAT(M3_, s), _), v))

#define TFBIG_KINIT(k0, k1, k2, k3, k4, k5, k6, k7, k8, t0, t1, t2) { \
		k8 = ((k0 ^ k1) ^ (k2 ^ k3)) ^ ((k4 ^ k5) ^ (k6 ^ k7)) \
			^ SPH_C64(0x1BD11BDAA9FC1A22); \
		t2 = t0 ^ t1; \
	}

#define TFBIG_ADDKEY(w0, w1, w2, w3, w4, w5, w6, w7, k, t, s) { \
		w0 = (w0 + SKBI(k, s, 0)); \
		w1 = (w1 + SKBI(k, s, 1)); \
		w2 = (w2 + SKBI(k, s, 2)); \
		w3 = (w3 + SKBI(k, s, 3)); \
		w4 = (w4 + SKBI(k, s, 4)); \
		w5 = (w5 + SKBI(k, s, 5) + SKBT(t, s, 0)); \
		w6 = (w6 + SKBI(k, s, 6) + SKBT(t, s, 1)); \
		w7 = (w7 + SKBI(k, s, 7) + (uint64_t)s); \
	}

#define TFBIG_MIX(x0, x1, rc) { \
		x0 = x0 + x1; \
		x1 = ROTL64(x1, rc) ^ x0; \
	}

#define TFBIG_MIX8(w0, w1, w2, w3, w4, w5, w6, w7, rc0, rc1, rc2, rc3) { \
		TFBIG_MIX(w0, w1, rc0); \
		TFBIG_MIX(w2, w3, rc1); \
		TFBIG_MIX(w4, w5, rc2); \
		TFBIG_MIX(w6, w7, rc3); \
	}

#define TFBIG_4e(s)  { \
		TFBIG_ADDKEY(p[0], p[1], p[2], p[3], p[4], p[5], p[6], p[7], h, t, s); \
		TFBIG_MIX8(p[0], p[1], p[2], p[3], p[4], p[5], p[6], p[7], 46, 36, 19, 37); \
		TFBIG_MIX8(p[2], p[1], p[4], p[7], p[6], p[5], p[0], p[3], 33, 27, 14, 42); \
		TFBIG_MIX8(p[4], p[1], p[6], p[3], p[0], p[5], p[2], p[7], 17, 49, 36, 39); \
		TFBIG_MIX8(p[6], p[1], p[0], p[7], p[2], p[5], p[4], p[3], 44,  9, 54, 56); \
	}

#define TFBIG_4o(s)  { \
		TFBIG_ADDKEY(p[0], p[1], p[2], p[3], p[4], p[5], p[6], p[7], h, t, s); \
		TFBIG_MIX8(p[0], p[1], p[2], p[3], p[4], p[5], p[6], p[7], 39, 30, 34, 24); \
		TFBIG_MIX8(p[2], p[1], p[4], p[7], p[6], p[5], p[0], p[3], 13, 50, 10, 17); \
		TFBIG_MIX8(p[4], p[1], p[6], p[3], p[0], p[5], p[2], p[7], 25, 29, 39, 43); \
		TFBIG_MIX8(p[6], p[1], p[0], p[7], p[2], p[5], p[4], p[3],  8, 35, 56, 22); \
	}
///////////////////////////
#define uTFBIG_KINIT(k0, k1, k2, k3, k4, k5, k6, k7, k8, t0, t1, t2) { \
		k8 = ((k0 ^ k1) ^ (k2 ^ k3)) ^ ((k4 ^ k5) ^ (k6 ^ k7)) \
			^ vectorize(0x1BD11BDAA9FC1A22); \
		t2 = t0 ^ t1; \
		}

#define uTFBIG_ADDKEY(w0, w1, w2, w3, w4, w5, w6, w7, k, t, s) { \
		w0 = (w0 + SKBI(k, s, 0)); \
		w1 = (w1 + SKBI(k, s, 1)); \
		w2 = (w2 + SKBI(k, s, 2)); \
		w3 = (w3 + SKBI(k, s, 3)); \
		w4 = (w4 + SKBI(k, s, 4)); \
		w5 = (w5 + SKBI(k, s, 5) + SKBT(t, s, 0)); \
		w6 = (w6 + SKBI(k, s, 6) + SKBT(t, s, 1)); \
		w7 = (w7 + SKBI(k, s, 7) + vectorize(s)); \
		}

#define uTFBIG_MIX(x0, x1, rc) { \
		x0 = x0 + x1; \
		x1 = ROL2(x1, rc) ^ x0; \
		}

#define uTFBIG_MIX8(w0, w1, w2, w3, w4, w5, w6, w7, rc0, rc1, rc2, rc3) { \
		uTFBIG_MIX(w0, w1, rc0); \
		uTFBIG_MIX(w2, w3, rc1); \
		uTFBIG_MIX(w4, w5, rc2); \
		uTFBIG_MIX(w6, w7, rc3); \
		}

#define uTFBIG_4e(s)  { \
		uTFBIG_ADDKEY(p[0], p[1], p[2], p[3], p[4], p[5], p[6], p[7], h, t, s); \
		uTFBIG_MIX8(p[0], p[1], p[2], p[3], p[4], p[5], p[6], p[7], 46, 36, 19, 37); \
		uTFBIG_MIX8(p[2], p[1], p[4], p[7], p[6], p[5], p[0], p[3], 33, 27, 14, 42); \
		uTFBIG_MIX8(p[4], p[1], p[6], p[3], p[0], p[5], p[2], p[7], 17, 49, 36, 39); \
		uTFBIG_MIX8(p[6], p[1], p[0], p[7], p[2], p[5], p[4], p[3], 44,  9, 54, 56); \
		}

#define uTFBIG_4o(s)  { \
		uTFBIG_ADDKEY(p[0], p[1], p[2], p[3], p[4], p[5], p[6], p[7], h, t, s); \
		uTFBIG_MIX8(p[0], p[1], p[2], p[3], p[4], p[5], p[6], p[7], 39, 30, 34, 24); \
		uTFBIG_MIX8(p[2], p[1], p[4], p[7], p[6], p[5], p[0], p[3], 13, 50, 10, 17); \
		uTFBIG_MIX8(p[4], p[1], p[6], p[3], p[0], p[5], p[2], p[7], 25, 29, 39, 43); \
		uTFBIG_MIX8(p[6], p[1], p[0], p[7], p[2], p[5], p[4], p[3],  8, 35, 56, 22); \
		}
//////////////////////////////////////////
static __constant__ uint64_t d_constMem[8];
static uint64_t h_constMem[8] = {
	SPH_C64(0x4903ADFF749C51CE),
	SPH_C64(0x0D95DE399746DF03),
	SPH_C64(0x8FD1934127C79BCE),
	SPH_C64(0x9A255629FF352CB1),
	SPH_C64(0x5DB62599DF6CA7B0),
	SPH_C64(0xEABE394CA9D5C3F4),
	SPH_C64(0x991112C71A75B523),
	SPH_C64(0xAE18A40B660FCC33) };

static __constant__ uint2 t12[6] = 
{
	{ 0x40, 0x0 }, { 0, 0xf0000000 }, {0x40,0xf0000000},
	{ 0x8, 0x0 }, { 0, 0xff000000 }, {0x8,0xff000000}
};


static __device__ __forceinline__ void tfbig_addkey_uint2(uint2 &w0, uint2 &w1, uint2 &w2, uint2 &w3, uint2 &w4, uint2 &w5, uint2 &w6, uint2 &w7,
	uint2 *k, uint2 *t, int s0, int s1, int s2, int s3, int s4, int s5, int s6, int s7, int t0, int t1, int s)
{
	w0 += k[s0];
	w1 += k[s1];
	w2 += k[s2];
	w3 += k[s3];
	w4 += k[s4];
	w5 += k[s5] + t[t0];
	w6 += k[s6] + t[t1];
	w7 += k[s7] + vectorize(s);
}

static __device__ __forceinline__ void tfbig_4e_uint2(uint2 &w0, uint2 &w1, uint2 &w2, uint2 &w3, uint2 &w4, uint2 &w5, uint2 &w6, uint2 &w7)
{

	w0 += w1;
	w2 += w3;
	w4 += w5;
	w6 += w7;
	w1 = ROL2(w1, 46);
	w3 = ROL2(w3, 36);
	w5 = ROL2(w5, 19);
	w7 = ROL2(w7, 37);
	w1 ^= w0;
	w3 ^= w2;
	w5 ^= w4;
	w7 ^= w6;


	w0 += w3;
	w2 += w1;
	w4 += w7;
	w6 += w5;
	w1 = ROL2(w1, 33);
	w7 = ROL2(w7, 27);
	w5 = ROL2(w5, 14);
	w3 = ROL2(w3, 42);
	w1 ^= w2;
	w3 ^= w0;
	w5 ^= w6;
	w7 ^= w4;

	w0 += w5;
	w2 += w7;
	w4 += w1;
	w6 += w3;
	w1 = ROL2(w1, 17);
	w3 = ROL2(w3, 49);
	w5 = ROL2(w5, 36);
	w7 = ROL2(w7, 39);
	w1 ^= w4;
	w3 ^= w6;
	w5 ^= w0;
	w7 ^= w2;

	w0 += w7;
	w2 += w5;
	w4 += w3;
	w6 += w1;
	w1 = ROL2(w1, 44);
	w7 = ROL2(w7, 9);
	w5 = ROL2(w5, 54);
	w3 = ROL2(w3, 56);
	w1 ^= w6;
	w3 ^= w4;
	w5 ^= w2;
	w7 ^= w0;



}

static __device__ __forceinline__ void tfbig_4o_uint2(uint2 &w0, uint2 &w1, uint2 &w2, uint2 &w3, uint2 &w4, uint2 &w5, uint2 &w6, uint2 &w7)
{

	w0 += w1;
	w2 += w3;
	w4 += w5;
	w6 += w7;
	w1 = ROL2(w1, 39);
	w3 = ROL2(w3, 30);
	w5 = ROL2(w5, 34);
	w7 = ROL2(w7, 24);
	w1 ^= w0;
	w3 ^= w2;
	w5 ^= w4;
	w7 ^= w6;


	w0 += w3;
	w2 += w1;
	w4 += w7;
	w6 += w5;
	w1 = ROL2(w1, 13);
	w7 = ROL2(w7, 50);
	w5 = ROL2(w5, 10);
	w3 = ROL2(w3, 17);
	w1 ^= w2;
	w3 ^= w0;
	w5 ^= w6;
	w7 ^= w4;

	w0 += w5;
	w2 += w7;
	w4 += w1;
	w6 += w3;
	w1 = ROL2(w1, 25);
	w3 = ROL2(w3, 29);
	w5 = ROL2(w5, 39);
	w7 = ROL2(w7, 43);
	w1 ^= w4;
	w3 ^= w6;
	w5 ^= w0;
	w7 ^= w2;


	w0 += w7;
	w2 += w5;
	w4 += w3;
	w6 += w1;
	w1 = ROL2(w1, 8);
	w7 = ROL2(w7, 35);
	w5 = ROL2(w5, 56);
	w3 = ROL2(w3, 22);
	w1 ^= w6;
	w3 ^= w4;
	w5 ^= w2;
	w7 ^= w0;

}


static __device__ __forceinline__ void tfbig_4e(uint64_t &w0, uint64_t &w1, uint64_t &w2, uint64_t &w3, uint64_t &w4, uint64_t &w5, uint64_t &w6, uint64_t &w7)
{

	w0 += w1;
	w2 += w3;
	w4 += w5;
	w6 += w7;
	w1 = ROTL64(w1, 46);
	w3 = ROTL64(w3, 36);
	w5 = ROTL64(w5, 19);
	w7 = ROTL64(w7, 37);
	w1 = xor1(w1, w0);
	w3 = xor1(w3, w2);
	w5 = xor1(w5, w4);
	w7 = xor1(w7, w6);


	w0 += w3;
	w2 += w1;
	w4 += w7;
	w6 += w5;
	w1 = ROTL64(w1, 33);
	w7 = ROTL64(w7, 27);
	w5 = ROTL64(w5, 14);
	w3 = ROTL64(w3, 42);
	w1 = xor1(w1, w2);
	w3 = xor1(w3, w0);
	w5 = xor1(w5, w6);
	w7 = xor1(w7, w4);

	w0 += w5;
	w2 += w7;
	w4 += w1;
	w6 += w3;
	w1 = ROTL64(w1, 17);
	w3 = ROTL64(w3, 49);
	w5 = ROTL64(w5, 36);
	w7 = ROTL64(w7, 39);
	w1 = xor1(w1, w4);
	w3 = xor1(w3, w6);
	w5 = xor1(w5, w0);
	w7 = xor1(w7, w2);

	w0 += w7;
	w2 += w5;
	w4 += w3;
	w6 += w1;
	w1 = ROTL64(w1, 44);
	w7 = ROTL64(w7, 9);
	w5 = ROTL64(w5, 54);
	w3 = ROTL64(w3, 56);
	w1 = xor1(w1, w6);
	w3 = xor1(w3, w4);
	w5 = xor1(w5, w2);
	w7 = xor1(w7, w0);



}

static __device__ __forceinline__ void tfbig_addkey(uint64_t &w0, uint64_t &w1, uint64_t &w2, uint64_t &w3, uint64_t &w4, uint64_t &w5, uint64_t &w6, uint64_t &w7, uint64_t *k, uint64_t* t,
	int s0, int s1, int s2, int s3, int s4, int s5, int s6, int s7, int t0, int t1, int s)
{
	w0 += k[s0];
	w1 += k[s1];
	w2 += k[s2];
	w3 += k[s3];
	w4 += k[s4];
	w5 += k[s5] + t[t0];
	w6 += k[s6] + t[t1];
	w7 += k[s7] + s;
}


static __device__ __forceinline__ void tfbig_4o(uint64_t &w0, uint64_t &w1, uint64_t &w2, uint64_t &w3, uint64_t &w4, uint64_t &w5, uint64_t &w6, uint64_t &w7)
{

	w0 += w1;
	w2 += w3;
	w4 += w5;
	w6 += w7;
	w1 = ROTL64(w1, 39);
	w3 = ROTL64(w3, 30);
	w5 = ROTL64(w5, 34);
	w7 = ROTL64(w7, 24);
	w1 = xor1(w1, w0);
	w3 = xor1(w3, w2);
	w5 = xor1(w5, w4);
	w7 = xor1(w7, w6);

	w0 += w3;
	w2 += w1;
	w4 += w7;
	w6 += w5;
	w1 = ROTL64(w1, 13);
	w7 = ROTL64(w7, 50);
	w5 = ROTL64(w5, 10);
	w3 = ROTL64(w3, 17);
	w1 = xor1(w1, w2);
	w3 = xor1(w3, w0);
	w5 = xor1(w5, w6);
	w7 = xor1(w7, w4);

	w0 += w5;
	w2 += w7;
	w4 += w1;
	w6 += w3;
	w1 = ROTL64(w1, 25);
	w3 = ROTL64(w3, 29);
	w5 = ROTL64(w5, 39);
	w7 = ROTL64(w7, 43);
	w1 = xor1(w1, w4);
	w3 = xor1(w3, w6);
	w5 = xor1(w5, w0);
	w7 = xor1(w7, w2);


	w0 += w7;
	w2 += w5;
	w4 += w3;
	w6 += w1;
	w1 = ROTL64(w1, 8);
	w7 = ROTL64(w7, 35);
	w5 = ROTL64(w5, 56);
	w3 = ROTL64(w3, 22);
	w1 = xor1(w1, w6);
	w3 = xor1(w3, w4);
	w5 = xor1(w5, w2);
	w7 = xor1(w7, w0);

}




__global__ void quark_skein512_gpu_hash_64(int threads, uint32_t startNounce, uint64_t *g_hash, uint32_t *g_nonceVector)
{
	int thread = (blockDim.x * blockIdx.x + threadIdx.x);
	if (thread < threads)
	{
		// Skein
		uint64_t p[8];
		uint64_t h0, h1, h2, h3, h4, h5, h6, h7, h8;
		uint64_t t0, t1, t2;

		uint32_t nounce = (g_nonceVector != NULL) ? g_nonceVector[thread] : (startNounce + thread);

		int hashPosition = nounce - startNounce;
		uint64_t *inpHash = &g_hash[8 * hashPosition];

		// Initialisierung
		h0 = d_constMem[0];
		h1 = d_constMem[1];
		h2 = d_constMem[2];
		h3 = d_constMem[3];
		h4 = d_constMem[4];
		h5 = d_constMem[5];
		h6 = d_constMem[6];
		h7 = d_constMem[7];

		// 1. Runde -> etype = 480, ptr = 64, bcount = 0, data = msg		
#pragma unroll 8
		for(int i=0;i<8;i++)
			p[i] = inpHash[i];

		t0 = 64; // ptr
		t1 = 480ull << 55; // etype
		TFBIG_KINIT(h0, h1, h2, h3, h4, h5, h6, h7, h8, t0, t1, t2);
		TFBIG_4e(0);
		TFBIG_4o(1);
		TFBIG_4e(2);
		TFBIG_4o(3);
		TFBIG_4e(4);
		TFBIG_4o(5);
		TFBIG_4e(6);
		TFBIG_4o(7);
		TFBIG_4e(8);
		TFBIG_4o(9);
		TFBIG_4e(10);
		TFBIG_4o(11);
		TFBIG_4e(12);
		TFBIG_4o(13);
		TFBIG_4e(14);
		TFBIG_4o(15);
		TFBIG_4e(16);
		TFBIG_4o(17);
		TFBIG_ADDKEY(p[0], p[1], p[2], p[3], p[4], p[5], p[6], p[7], h, t, 18);

		h0 = inpHash[0] ^ p[0];
		h1 = inpHash[1] ^ p[1];
		h2 = inpHash[2] ^ p[2];
		h3 = inpHash[3] ^ p[3];
		h4 = inpHash[4] ^ p[4];
		h5 = inpHash[5] ^ p[5];
		h6 = inpHash[6] ^ p[6];
		h7 = inpHash[7] ^ p[7];

		// 2. Runde -> etype = 510, ptr = 8, bcount = 0, data = 0
#pragma unroll 8
		for(int i=0;i<8;i++)
			p[i] = 0;

		t0 = 8; // ptr
		t1 = 510ull << 55; // etype
		TFBIG_KINIT(h0, h1, h2, h3, h4, h5, h6, h7, h8, t0, t1, t2);
		TFBIG_4e(0);
		TFBIG_4o(1);
		TFBIG_4e(2);
		TFBIG_4o(3);
		TFBIG_4e(4);
		TFBIG_4o(5);
		TFBIG_4e(6);
		TFBIG_4o(7);
		TFBIG_4e(8);
		TFBIG_4o(9);
		TFBIG_4e(10);
		TFBIG_4o(11);
		TFBIG_4e(12);
		TFBIG_4o(13);
		TFBIG_4e(14);
		TFBIG_4o(15);
		TFBIG_4e(16);
		TFBIG_4o(17);
		TFBIG_ADDKEY(p[0], p[1], p[2], p[3], p[4], p[5], p[6], p[7], h, t, 18);

		// fertig
		uint64_t *outpHash = &g_hash[8 * hashPosition];

#pragma unroll 8
		for(int i=0;i<8;i++)
			outpHash[i] = p[i];
	}
}


__global__ void ziftr_skein512_gpu_hash_64(int threads, uint32_t startNounce, uint64_t *g_hash, uint8_t *d_test,uint32_t table)
{
	int thread = (blockDim.x * blockIdx.x + threadIdx.x);
	if (thread < threads)
	{

			if ((d_test + 4 * thread)[table & (~0xFFFF0000)] == ((table & (~0x0000FFFF)) >> 16)) {


		// Skein
		uint64_t p[8];
		uint64_t h0, h1, h2, h3, h4, h5, h6, h7, h8;
		uint64_t t0, t1, t2;

		uint32_t nounce = startNounce + thread;

		int hashPosition = nounce - startNounce;
		uint64_t *inpHash = &g_hash[8 * hashPosition];
		
		// Initialisierung
		h0 = d_constMem[0];
		h1 = d_constMem[1];
		h2 = d_constMem[2];
		h3 = d_constMem[3];
		h4 = d_constMem[4];
		h5 = d_constMem[5];
		h6 = d_constMem[6];
		h7 = d_constMem[7];

		// 1. Runde -> etype = 480, ptr = 64, bcount = 0, data = msg		
#pragma unroll 8
		for (int i = 0; i<8; i++)
			p[i] = inpHash[i];

		t0 = 64; // ptr
		t1 = 480ull << 55; // etype
		TFBIG_KINIT(h0, h1, h2, h3, h4, h5, h6, h7, h8, t0, t1, t2);
		TFBIG_4e(0);
		TFBIG_4o(1);
		TFBIG_4e(2);
		TFBIG_4o(3);
		TFBIG_4e(4);
		TFBIG_4o(5);
		TFBIG_4e(6);
		TFBIG_4o(7);
		TFBIG_4e(8);
		TFBIG_4o(9);
		TFBIG_4e(10);
		TFBIG_4o(11);
		TFBIG_4e(12);
		TFBIG_4o(13);
		TFBIG_4e(14);
		TFBIG_4o(15);
		TFBIG_4e(16);
		TFBIG_4o(17);
		TFBIG_ADDKEY(p[0], p[1], p[2], p[3], p[4], p[5], p[6], p[7], h, t, 18);

		h0 = inpHash[0] ^ p[0];
		h1 = inpHash[1] ^ p[1];
		h2 = inpHash[2] ^ p[2];
		h3 = inpHash[3] ^ p[3];
		h4 = inpHash[4] ^ p[4];
		h5 = inpHash[5] ^ p[5];
		h6 = inpHash[6] ^ p[6];
		h7 = inpHash[7] ^ p[7];

		// 2. Runde -> etype = 510, ptr = 8, bcount = 0, data = 0
#pragma unroll 8
		for (int i = 0; i<8; i++)
			p[i] = 0;

		t0 = 8; // ptr
		t1 = 510ull << 55; // etype
		TFBIG_KINIT(h0, h1, h2, h3, h4, h5, h6, h7, h8, t0, t1, t2);
		TFBIG_4e(0);
		TFBIG_4o(1);
		TFBIG_4e(2);
		TFBIG_4o(3);
		TFBIG_4e(4);
		TFBIG_4o(5);
		TFBIG_4e(6);
		TFBIG_4o(7);
		TFBIG_4e(8);
		TFBIG_4o(9);
		TFBIG_4e(10);
		TFBIG_4o(11);
		TFBIG_4e(12);
		TFBIG_4o(13);
		TFBIG_4e(14);
		TFBIG_4o(15);
		TFBIG_4e(16);
		TFBIG_4o(17);
		TFBIG_ADDKEY(p[0], p[1], p[2], p[3], p[4], p[5], p[6], p[7], h, t, 18);

		// fertig
		uint64_t *outpHash = &g_hash[8 * hashPosition];

#pragma unroll 8
		for (int i = 0; i<8; i++)
			outpHash[i] = p[i];

		
       } // table
	} // thread
}

__global__ void ziftr_skein512uint2_gpu_hash_64(int threads, uint32_t startNounce, uint64_t *g_hash, uint8_t *d_test, uint32_t table)
{
	int thread = (blockDim.x * blockIdx.x + threadIdx.x);
	if (thread < threads)
	{

		if ((d_test + 4 * thread)[table & (~0xFFFF0000)] == ((table & (~0x0000FFFF)) >> 16)) {


			// Skein
			uint2 p[8];
			uint2 h0, h1, h2, h3, h4, h5, h6, h7, h8;
			uint2 t0, t1, t2;

			uint32_t nounce = startNounce + thread;

			int hashPosition = nounce - startNounce;
			uint64_t *inpHash = &g_hash[8 * hashPosition];

			// Initialisierung
			h0 = vectorize(d_constMem[0]);
			h1 = vectorize(d_constMem[1]);
			h2 = vectorize(d_constMem[2]);
			h3 = vectorize(d_constMem[3]);
			h4 = vectorize(d_constMem[4]);
			h5 = vectorize(d_constMem[5]);
			h6 = vectorize(d_constMem[6]);
			h7 = vectorize(d_constMem[7]);

			// 1. Runde -> etype = 480, ptr = 64, bcount = 0, data = msg		
#pragma unroll 8
			for (int i = 0; i<8; i++)
				p[i] = vectorize(inpHash[i]);

			t0 = vectorize(64); // ptr
			t1 = vectorize(480ull << 55); // etype
			uTFBIG_KINIT(h0, h1, h2, h3, h4, h5, h6, h7, h8, t0, t1, t2);
			uTFBIG_4e(0);
			uTFBIG_4o(1);
			uTFBIG_4e(2);
			uTFBIG_4o(3);
			uTFBIG_4e(4);
			uTFBIG_4o(5);
			uTFBIG_4e(6);
			uTFBIG_4o(7);
			uTFBIG_4e(8);
			uTFBIG_4o(9);
			uTFBIG_4e(10);
			uTFBIG_4o(11);
			uTFBIG_4e(12);
			uTFBIG_4o(13);
			uTFBIG_4e(14);
			uTFBIG_4o(15);
			uTFBIG_4e(16);
			uTFBIG_4o(17);
			uTFBIG_ADDKEY(p[0], p[1], p[2], p[3], p[4], p[5], p[6], p[7], h, t, 18);

			h0 = vectorize(inpHash[0]) ^ p[0];
			h1 = vectorize(inpHash[1]) ^ p[1];
			h2 = vectorize(inpHash[2]) ^ p[2];
			h3 = vectorize(inpHash[3]) ^ p[3];
			h4 = vectorize(inpHash[4]) ^ p[4];
			h5 = vectorize(inpHash[5]) ^ p[5];
			h6 = vectorize(inpHash[6]) ^ p[6];
			h7 = vectorize(inpHash[7]) ^ p[7];

			// 2. Runde -> etype = 510, ptr = 8, bcount = 0, data = 0
#pragma unroll 8
			for (int i = 0; i<8; i++)
				p[i] = make_uint2(0,0);

			t0 = vectorize(8); // ptr
			t1 = vectorize(510ull << 55); // etype
			uTFBIG_KINIT(h0, h1, h2, h3, h4, h5, h6, h7, h8, t0, t1, t2);
			uTFBIG_4e(0);
			uTFBIG_4o(1);
			uTFBIG_4e(2);
			uTFBIG_4o(3);
			uTFBIG_4e(4);
			uTFBIG_4o(5);
			uTFBIG_4e(6);
			uTFBIG_4o(7);
			uTFBIG_4e(8);
			uTFBIG_4o(9);
			uTFBIG_4e(10);
			uTFBIG_4o(11);
			uTFBIG_4e(12);
			uTFBIG_4o(13);
			uTFBIG_4e(14);
			uTFBIG_4o(15);
			uTFBIG_4e(16);
			uTFBIG_4o(17);
			uTFBIG_ADDKEY(p[0], p[1], p[2], p[3], p[4], p[5], p[6], p[7], h, t, 18);

			// fertig
			uint64_t *outpHash = &g_hash[8 * hashPosition];

#pragma unroll 8
			for (int i = 0; i<8; i++)
				outpHash[i] = devectorize(p[i]);


		} // table
	} // thread
}

// Setup-Funktionen
__host__ void quark_skein512_cpu_init(int thr_id, int threads)
{
	// nix zu tun ;-)
	hipMemcpyToSymbol(HIP_SYMBOL( d_constMem),
                        h_constMem,
                        sizeof(h_constMem),
                        0, hipMemcpyHostToDevice);
}

__host__ void quark_skein512_cpu_hash_64(int thr_id, int threads, uint32_t startNounce, uint32_t *d_nonceVector, uint32_t *d_hash, int order)
{
	const int threadsperblock = 256;

	// berechne wie viele Thread Blocks wir brauchen
	dim3 grid((threads + threadsperblock-1)/threadsperblock);
	dim3 block(threadsperblock);

	// Gr��e des dynamischen Shared Memory Bereichs
	size_t shared_size = 0;

	quark_skein512_gpu_hash_64<<<grid, block, shared_size>>>(threads, startNounce, (uint64_t*)d_hash, d_nonceVector);

	// Strategisches Sleep Kommando zur Senkung der CPU Last
	MyStreamSynchronize(NULL, order, thr_id);
}

__host__ void ziftr_skein512_cpu_hash_64(int thr_id, int threads, uint32_t startNounce, uint32_t *d_hash, uint32_t* d_test,uint32_t table,int order)
{
	const int threadsperblock = 256;

	// berechne wie viele Thread Blocks wir brauchen
	dim3 grid((threads + threadsperblock - 1) / threadsperblock);
	dim3 block(threadsperblock);

	// Gr��e des dynamischen Shared Memory Bereichs
	size_t shared_size = 0;
	if (compute_version[thr_id]<50) {
	ziftr_skein512_gpu_hash_64 << <grid, block, shared_size >> >(threads, startNounce, (uint64_t*)d_hash, (uint8_t*)d_test, table);
	} else {
	ziftr_skein512uint2_gpu_hash_64 << <grid, block, shared_size >> >(threads, startNounce, (uint64_t*)d_hash, (uint8_t*)d_test,table);
    }
	// Strategisches Sleep Kommando zur Senkung der CPU Last
	MyStreamSynchronize(NULL, order, thr_id);
}
