#include "hip/hip_runtime.h"

#include <hip/hip_runtime.h>
#include "hip/hip_runtime.h"
#include ""


#include <stdio.h>
#include <stdint.h>
#include <memory.h>


extern hipError_t MyStreamSynchronize(hipStream_t stream, int situation, int thr_id);
extern int compute_version[8];

#include "cuda_helper.h"
static __constant__ uint64_t stateo[25];
static __constant__ uint64_t RC[24];
static const uint64_t cpu_RC[24] = {
    0x0000000000000001ull, 0x0000000000008082ull,
    0x800000000000808aull, 0x8000000080008000ull,
    0x000000000000808bull, 0x0000000080000001ull,
    0x8000000080008081ull, 0x8000000000008009ull,
    0x000000000000008aull, 0x0000000000000088ull,
    0x0000000080008009ull, 0x000000008000000aull,
    0x000000008000808bull, 0x800000000000008bull,
    0x8000000000008089ull, 0x8000000000008003ull,
    0x8000000000008002ull, 0x8000000000000080ull,
    0x000000000000800aull, 0x800000008000000aull,
    0x8000000080008081ull, 0x8000000000008080ull,
    0x0000000080000001ull, 0x8000000080008008ull
};

static __constant__ uchar4 arrOrder[24] =
{
	{ 4, 1, 2, 3 },
	{ 4, 1, 3, 2 },
	{ 4, 2, 1, 3 },
	{ 4, 2, 3, 1 },
	{ 4, 3, 1, 2 },
	{ 4, 3, 2, 1 },
	{ 1, 4, 2, 3 },
	{ 1, 4, 3, 2 },
	{ 1, 2, 4, 3 },
	{ 1, 2, 3, 4 },
	{ 1, 3, 4, 2 },
	{ 1, 3, 2, 4 },
	{ 2, 4, 1, 3 },
	{ 2, 4, 3, 1 },
	{ 2, 1, 4, 3 },
	{ 2, 1, 3, 4 },
	{ 2, 3, 4, 1 },
	{ 2, 3, 1, 4 },
	{ 3, 4, 1, 2 },
	{ 3, 4, 2, 1 },
	{ 3, 1, 4, 2 },
	{ 3, 1, 2, 4 },
	{ 3, 2, 4, 1 },
	{ 3, 2, 1, 4 }
};
static __device__ __forceinline__ void keccak_block(uint64_t *s, const uint64_t *keccak_round_constants) {
    size_t i;
    uint64_t t[5], u[5], v, w;

    /* absorb input */    
    
//#pragma unroll 24
    for (i = 0; i < 24; i++) {
        /* theta: c = a[0,i] ^ a[1,i] ^ .. a[4,i] */
		
        t[0] = s[0] ^ s[5] ^ s[10] ^ s[15] ^ s[20];
        t[1] = s[1] ^ s[6] ^ s[11] ^ s[16] ^ s[21];
        t[2] = s[2] ^ s[7] ^ s[12] ^ s[17] ^ s[22];
        t[3] = s[3] ^ s[8] ^ s[13] ^ s[18] ^ s[23];
        t[4] = s[4] ^ s[9] ^ s[14] ^ s[19] ^ s[24]; 
		 
        /* theta: d[i] = c[i+4] ^ rotl(c[i+1],1) */
		
		uint64_t temp0,temp1,temp2,temp3,temp4;
        temp0 = ROTL64(t[0], 1);
		temp1 = ROTL64(t[1], 1);
		temp2 = ROTL64(t[2], 1);
		temp3 = ROTL64(t[3], 1);
		temp4 = ROTL64(t[4], 1);
		u[0] = xor1(t[4],temp1);
        u[1] = xor1(t[0],temp2);
        u[2] = xor1(t[1],temp3);
        u[3] = xor1(t[2],temp4);
        u[4] = xor1(t[3],temp0);
		
        /* theta: a[0,i], a[1,i], .. a[4,i] ^= d[i] */
        s[0] ^= u[0]; s[5] ^= u[0]; s[10] ^= u[0]; s[15] ^= u[0]; s[20] ^= u[0];
        s[1] ^= u[1]; s[6] ^= u[1]; s[11] ^= u[1]; s[16] ^= u[1]; s[21] ^= u[1];
        s[2] ^= u[2]; s[7] ^= u[2]; s[12] ^= u[2]; s[17] ^= u[2]; s[22] ^= u[2];
        s[3] ^= u[3]; s[8] ^= u[3]; s[13] ^= u[3]; s[18] ^= u[3]; s[23] ^= u[3];
        s[4] ^= u[4]; s[9] ^= u[4]; s[14] ^= u[4]; s[19] ^= u[4]; s[24] ^= u[4];

        /* rho pi: b[..] = rotl(a[..], ..) */
        v = s[ 1];
        s[ 1] = ROTL64(s[ 6], 44);
        s[ 6] = ROTL64(s[ 9], 20);
        s[ 9] = ROTL64(s[22], 61);
        s[22] = ROTL64(s[14], 39);
        s[14] = ROTL64(s[20], 18);
        s[20] = ROTL64(s[ 2], 62);
        s[ 2] = ROTL64(s[12], 43);
        s[12] = ROTL64(s[13], 25);
        s[13] = ROTL64(s[19],  8);
        s[19] = ROTL64(s[23], 56);
        s[23] = ROTL64(s[15], 41);
        s[15] = ROTL64(s[ 4], 27);
        s[ 4] = ROTL64(s[24], 14);
        s[24] = ROTL64(s[21],  2);
        s[21] = ROTL64(s[ 8], 55);
        s[ 8] = ROTL64(s[16], 45);
        s[16] = ROTL64(s[ 5], 36);
        s[ 5] = ROTL64(s[ 3], 28);
        s[ 3] = ROTL64(s[18], 21);
        s[18] = ROTL64(s[17], 15);
        s[17] = ROTL64(s[11], 10);
        s[11] = ROTL64(s[ 7],  6);
        s[ 7] = ROTL64(s[10],  3);
        s[10] = ROTL64(    v,  1);

        /* chi: a[i,j] ^= ~b[i,j+1] & b[i,j+2] */		

		v = s[ 0]; w = s[ 1]; 
		s[ 0] ^= (~w) & s[ 2]; 
		s[ 1] ^= (~s[ 2]) & s[ 3]; 
		s[ 2] ^= (~s[ 3]) & s[ 4]; 
		s[ 3] ^= (~s[ 4]) & v; 
		s[ 4] ^= (~v) & w;
		v = s[ 5]; w = s[ 6];
		s[ 5] ^= (~w) & s[ 7];
		s[ 6] ^= (~s[ 7]) & s[ 8];
		s[ 7] ^= (~s[ 8]) & s[ 9];
		s[ 8] ^= (~s[ 9]) & v;
		s[ 9] ^= (~v) & w;
        v = s[10]; w = s[11];
		s[10] ^= (~w) & s[12];
		s[11] ^= (~s[12]) & s[13];
		s[12] ^= (~s[13]) & s[14];
		s[13] ^= (~s[14]) & v;
		s[14] ^= (~v) & w;
        v = s[15]; w = s[16];
		s[15] ^= (~w) & s[17];
		s[16] ^= (~s[17]) & s[18];
		s[17] ^= (~s[18]) & s[19];
		s[18] ^= (~s[19]) & v;
		s[19] ^= (~v) & w;
        v = s[20]; w = s[21];
		s[20] ^= (~w) & s[22];
		s[21] ^= (~s[22]) & s[23];
		s[22] ^= (~s[23]) & s[24];
		s[23] ^= (~s[24]) & v;
        s[24] ^= (~v) & w;
		
        /* iota: a[0,0] ^= round constant */
        s[0] ^= keccak_round_constants[i];
    }
}

static __device__ __forceinline__ void keccak_blockv4(uint2 *s, const uint64_t *keccak_round_constants) {
	size_t i;
	uint2 t[5], u[5], v, w;

	//    #pragma unroll
	for (i = 0; i < 24; i++) {
		/* theta: c = a[0,i] ^ a[1,i] ^ .. a[4,i] */
		t[0] = s[0] ^ s[5] ^ s[10] ^ s[15] ^ s[20];
		t[1] = s[1] ^ s[6] ^ s[11] ^ s[16] ^ s[21];
		t[2] = s[2] ^ s[7] ^ s[12] ^ s[17] ^ s[22];
		t[3] = s[3] ^ s[8] ^ s[13] ^ s[18] ^ s[23];
		t[4] = s[4] ^ s[9] ^ s[14] ^ s[19] ^ s[24];

		/* theta: d[i] = c[i+4] ^ rotl(c[i+1],1) */
		u[0] = t[4] ^ ROL2(t[1], 1);
		u[1] = t[0] ^ ROL2(t[2], 1);
		u[2] = t[1] ^ ROL2(t[3], 1);
		u[3] = t[2] ^ ROL2(t[4], 1);
		u[4] = t[3] ^ ROL2(t[0], 1);

		/* theta: a[0,i], a[1,i], .. a[4,i] ^= d[i] */
		s[0] ^= u[0]; s[5] ^= u[0]; s[10] ^= u[0]; s[15] ^= u[0]; s[20] ^= u[0];
		s[1] ^= u[1]; s[6] ^= u[1]; s[11] ^= u[1]; s[16] ^= u[1]; s[21] ^= u[1];
		s[2] ^= u[2]; s[7] ^= u[2]; s[12] ^= u[2]; s[17] ^= u[2]; s[22] ^= u[2];
		s[3] ^= u[3]; s[8] ^= u[3]; s[13] ^= u[3]; s[18] ^= u[3]; s[23] ^= u[3];
		s[4] ^= u[4]; s[9] ^= u[4]; s[14] ^= u[4]; s[19] ^= u[4]; s[24] ^= u[4];

		/* rho pi: b[..] = rotl(a[..], ..) */
		v = s[1];
		s[1] = ROL2(s[6], 44);
		s[6] = ROL2(s[9], 20);
		s[9] = ROL2(s[22], 61);
		s[22] = ROL2(s[14], 39);
		s[14] = ROL2(s[20], 18);
		s[20] = ROL2(s[2], 62);
		s[2] = ROL2(s[12], 43);
		s[12] = ROL2(s[13], 25);
		s[13] = ROL2(s[19], 8);
		s[19] = ROL2(s[23], 56);
		s[23] = ROL2(s[15], 41);
		s[15] = ROL2(s[4], 27);
		s[4] = ROL2(s[24], 14);
		s[24] = ROL2(s[21], 2);
		s[21] = ROL2(s[8], 55);
		s[8] = ROL2(s[16], 45);
		s[16] = ROL2(s[5], 36);
		s[5] = ROL2(s[3], 28);
		s[3] = ROL2(s[18], 21);
		s[18] = ROL2(s[17], 15);
		s[17] = ROL2(s[11], 10);
		s[11] = ROL2(s[7], 6);
		s[7] = ROL2(s[10], 3);
		s[10] = ROL2(v, 1);

		/* chi: a[i,j] ^= ~b[i,j+1] & b[i,j+2] */
		v = s[0]; w = s[1]; s[0] ^= (~w) & s[2]; s[1] ^= (~s[2]) & s[3]; s[2] ^= (~s[3]) & s[4]; s[3] ^= (~s[4]) & v; s[4] ^= (~v) & w;
		v = s[5]; w = s[6]; s[5] ^= (~w) & s[7]; s[6] ^= (~s[7]) & s[8]; s[7] ^= (~s[8]) & s[9]; s[8] ^= (~s[9]) & v; s[9] ^= (~v) & w;
		v = s[10]; w = s[11]; s[10] ^= (~w) & s[12]; s[11] ^= (~s[12]) & s[13]; s[12] ^= (~s[13]) & s[14]; s[13] ^= (~s[14]) & v; s[14] ^= (~v) & w;
		v = s[15]; w = s[16]; s[15] ^= (~w) & s[17]; s[16] ^= (~s[17]) & s[18]; s[17] ^= (~s[18]) & s[19]; s[18] ^= (~s[19]) & v; s[19] ^= (~v) & w;
		v = s[20]; w = s[21]; s[20] ^= (~w) & s[22]; s[21] ^= (~s[22]) & s[23]; s[22] ^= (~s[23]) & s[24]; s[23] ^= (~s[24]) & v; s[24] ^= (~v) & w;

		/* iota: a[0,0] ^= round constant */
		s[0] ^= vectorize(keccak_round_constants[i]);
	}
}


static __device__ __forceinline__ void keccak_blockv3(uint64_t *state, const uint64_t *keccak_round_constants)
{

	uint2 Aba, Abe, Abi, Abo, Abu;
	uint2 Aga, Age, Agi, Ago, Agu;
	uint2 Aka, Ake, Aki, Ako, Aku;
	uint2 Ama, Ame, Ami, Amo, Amu;
	uint2 Asa, Ase, Asi, Aso, Asu;
	uint2 BCa, BCe, BCi, BCo, BCu;
	uint2 Da, De, Di, Do, Du;
	uint2 Eba, Ebe, Ebi, Ebo, Ebu;
	uint2 Ega, Ege, Egi, Ego, Egu;
	uint2 Eka, Eke, Eki, Eko, Eku;
	uint2 Ema, Eme, Emi, Emo, Emu;
	uint2 Esa, Ese, Esi, Eso, Esu;
	Aba = vectorize(state[0]);
	Abe = vectorize(state[1]);
	Abi = vectorize(state[2]);
	Abo = vectorize(state[3]);
	Abu = vectorize(state[4]);
	Aga = vectorize(state[5]);
	Age = vectorize(state[6]);
	Agi = vectorize(state[7]);
	Ago = vectorize(state[8]);
	Agu = vectorize(state[9]);
	Aka = vectorize(state[10]);
	Ake = vectorize(state[11]);
	Aki = vectorize(state[12]);
	Ako = vectorize(state[13]);
	Aku = vectorize(state[14]);
	Ama = vectorize(state[15]);
	Ame = vectorize(state[16]);
	Ami = vectorize(state[17]);
	Amo = vectorize(state[18]);
	Amu = vectorize(state[19]);
	Asa = vectorize(state[20]);
	Ase = vectorize(state[21]);
	Asi = vectorize(state[22]);
	Aso = vectorize(state[23]);
	Asu = vectorize(state[24]);
    #pragma unroll 
	for (int round = 0; round < 24; round += 2)
	{
		//    int round =2;
		//    prepareTheta
		BCa = Aba^Aga^Aka^Ama^Asa;
		BCe = Abe^Age^Ake^Ame^Ase;
		BCi = Abi^Agi^Aki^Ami^Asi;
		BCo = Abo^Ago^Ako^Amo^Aso;
		BCu = Abu^Agu^Aku^Amu^Asu;

		//thetaRhoPiChiIotaPrepareTheta(round  , A, E)
		Da = BCu^ROL2(BCe, 1);
		De = BCa^ROL2(BCi, 1);
		Di = BCe^ROL2(BCo, 1);
		Do = BCi^ROL2(BCu, 1);
		Du = BCo^ROL2(BCa, 1);

		Aba ^= Da;
		BCa = Aba;
		Age ^= De;
		BCe = ROL2(Age, 44);
		Aki ^= Di;
		BCi = ROL2(Aki, 43);
		Amo ^= Do;
		BCo = ROL2(Amo, 21);
		Asu ^= Du;
		BCu = ROL2(Asu, 14);
		Eba = BCa ^ ((~BCe)&  BCi);
		Eba ^= vectorize(keccak_round_constants[round]);
		Ebe = BCe ^ ((~BCi)&  BCo);
		Ebi = BCi ^ ((~BCo)&  BCu);
		Ebo = BCo ^ ((~BCu)&  BCa);
		Ebu = BCu ^ ((~BCa)&  BCe);

		Abo ^= Do;
		BCa = ROL2(Abo, 28);
		Agu ^= Du;
		BCe = ROL2(Agu, 20);
		Aka ^= Da;
		BCi = ROL2(Aka, 3);
		Ame ^= De;
		BCo = ROL2(Ame, 45);
		Asi ^= Di;
		BCu = ROL2(Asi, 61);
		Ega = BCa ^ ((~BCe)&  BCi);
		Ege = BCe ^ ((~BCi)&  BCo);
		Egi = BCi ^ ((~BCo)&  BCu);
		Ego = BCo ^ ((~BCu)&  BCa);
		Egu = BCu ^ ((~BCa)&  BCe);

		Abe ^= De;
		BCa = ROL2(Abe, 1);
		Agi ^= Di;
		BCe = ROL2(Agi, 6);
		Ako ^= Do;
		BCi = ROL2(Ako, 25);
		Amu ^= Du;
		BCo = ROL2(Amu, 8);
		Asa ^= Da;
		BCu = ROL2(Asa, 18);
		Eka = BCa ^ ((~BCe)&  BCi);
		Eke = BCe ^ ((~BCi)&  BCo);
		Eki = BCi ^ ((~BCo)&  BCu);
		Eko = BCo ^ ((~BCu)&  BCa);
		Eku = BCu ^ ((~BCa)&  BCe);

		Abu ^= Du;
		BCa = ROL2(Abu, 27);
		Aga ^= Da;
		BCe = ROL2(Aga, 36);
		Ake ^= De;
		BCi = ROL2(Ake, 10);
		Ami ^= Di;
		BCo = ROL2(Ami, 15);
		Aso ^= Do;
		BCu = ROL2(Aso, 56);
		Ema = BCa ^ ((~BCe)&  BCi);
		Eme = BCe ^ ((~BCi)&  BCo);
		Emi = BCi ^ ((~BCo)&  BCu);
		Emo = BCo ^ ((~BCu)&  BCa);
		Emu = BCu ^ ((~BCa)&  BCe);

		Abi ^= Di;
		BCa = ROL2(Abi, 62);
		Ago ^= Do;
		BCe = ROL2(Ago, 55);
		Aku ^= Du;
		BCi = ROL2(Aku, 39);
		Ama ^= Da;
		BCo = ROL2(Ama, 41);
		Ase ^= De;
		BCu = ROL2(Ase, 2);
		Esa = BCa ^ ((~BCe)&  BCi);
		Ese = BCe ^ ((~BCi)&  BCo);
		Esi = BCi ^ ((~BCo)&  BCu);
		Eso = BCo ^ ((~BCu)&  BCa);
		Esu = BCu ^ ((~BCa)&  BCe);

		//    prepareTheta
		BCa = Eba^Ega^Eka^Ema^Esa;
		BCe = Ebe^Ege^Eke^Eme^Ese;
		BCi = Ebi^Egi^Eki^Emi^Esi;
		BCo = Ebo^Ego^Eko^Emo^Eso;
		BCu = Ebu^Egu^Eku^Emu^Esu;

		//thetaRhoPiChiIotaPrepareTheta(round+1, E, A)
		Da = BCu^ROL2(BCe, 1);
		De = BCa^ROL2(BCi, 1);
		Di = BCe^ROL2(BCo, 1);
		Do = BCi^ROL2(BCu, 1);
		Du = BCo^ROL2(BCa, 1);

		Eba ^= Da;
		BCa = Eba;
		Ege ^= De;
		BCe = ROL2(Ege, 44);
		Eki ^= Di;
		BCi = ROL2(Eki, 43);
		Emo ^= Do;
		BCo = ROL2(Emo, 21);
		Esu ^= Du;
		BCu = ROL2(Esu, 14);
		Aba = BCa ^ ((~BCe)&  BCi);
		Aba ^= vectorize(keccak_round_constants[round + 1]);
		Abe = BCe ^ ((~BCi)&  BCo);
		Abi = BCi ^ ((~BCo)&  BCu);
		Abo = BCo ^ ((~BCu)&  BCa);
		Abu = BCu ^ ((~BCa)&  BCe);

		Ebo ^= Do;
		BCa = ROL2(Ebo, 28);
		Egu ^= Du;
		BCe = ROL2(Egu, 20);
		Eka ^= Da;
		BCi = ROL2(Eka, 3);
		Eme ^= De;
		BCo = ROL2(Eme, 45);
		Esi ^= Di;
		BCu = ROL2(Esi, 61);
		Aga = BCa ^ ((~BCe)&  BCi);
		Age = BCe ^ ((~BCi)&  BCo);
		Agi = BCi ^ ((~BCo)&  BCu);
		Ago = BCo ^ ((~BCu)&  BCa);
		Agu = BCu ^ ((~BCa)&  BCe);

		Ebe ^= De;
		BCa = ROL2(Ebe, 1);
		Egi ^= Di;
		BCe = ROL2(Egi, 6);
		Eko ^= Do;
		BCi = ROL2(Eko, 25);
		Emu ^= Du;
		BCo = ROL2(Emu, 8);
		Esa ^= Da;
		BCu = ROL2(Esa, 18);
		Aka = BCa ^ ((~BCe)&  BCi);
		Ake = BCe ^ ((~BCi)&  BCo);
		Aki = BCi ^ ((~BCo)&  BCu);
		Ako = BCo ^ ((~BCu)&  BCa);
		Aku = BCu ^ ((~BCa)&  BCe);

		Ebu ^= Du;
		BCa = ROL2(Ebu, 27);
		Ega ^= Da;
		BCe = ROL2(Ega, 36);
		Eke ^= De;
		BCi = ROL2(Eke, 10);
		Emi ^= Di;
		BCo = ROL2(Emi, 15);
		Eso ^= Do;
		BCu = ROL2(Eso, 56);
		Ama = BCa ^ ((~BCe)&  BCi);
		Ame = BCe ^ ((~BCi)&  BCo);
		Ami = BCi ^ ((~BCo)&  BCu);
		Amo = BCo ^ ((~BCu)&  BCa);
		Amu = BCu ^ ((~BCa)&  BCe);

		Ebi ^= Di;
		BCa = ROL2(Ebi, 62);
		Ego ^= Do;
		BCe = ROL2(Ego, 55);
		Eku ^= Du;
		BCi = ROL2(Eku, 39);
		Ema ^= Da;
		BCo = ROL2(Ema, 41);
		Ese ^= De;
		BCu = ROL2(Ese, 2);
		Asa = BCa ^ ((~BCe)&  BCi);
		Ase = BCe ^ ((~BCi)&  BCo);
		Asi = BCi ^ ((~BCo)&  BCu);
		Aso = BCo ^ ((~BCu)&  BCa);
		Asu = BCu ^ ((~BCa)&  BCe);
	}



	state[0] = devectorize(Aba);
	state[1] = devectorize(Abe);
	state[2] = devectorize(Abi);
	state[3] = devectorize(Abo);
	state[4] = devectorize(Abu);
	state[5] = devectorize(Aga);
	state[6] = devectorize(Age);
	state[7] = devectorize(Agi);
	state[8] = devectorize(Ago);
	state[9] = devectorize(Agu);
	state[10] = devectorize(Aka);
	state[11] = devectorize(Ake);
	state[12] = devectorize(Aki);
	state[13] = devectorize(Ako);
	state[14] = devectorize(Aku);
	state[15] = devectorize(Ama);
	state[16] = devectorize(Ame);
	state[17] = devectorize(Ami);
	state[18] = devectorize(Amo);
	state[19] = devectorize(Amu);
	state[20] = devectorize(Asa);
	state[21] = devectorize(Ase);
	state[22] = devectorize(Asi);
	state[23] = devectorize(Aso);
	state[24] = devectorize(Asu);


	//	if (thread == 0) {for (int i=0;i<25;i++) {printf("i%d uint2 %08x %08x\n",i, LOWORD(state[i]), HIWORD(state[i])); }}
}



static __device__ __forceinline__ void keccak_blockv2(uint64_t *state, const uint64_t *keccak_round_constants) 
{



	{
		uint64_t Aba, Abe, Abi, Abo, Abu;
		uint64_t Aga, Age, Agi, Ago, Agu;
		uint64_t Aka, Ake, Aki, Ako, Aku;
		uint64_t Ama, Ame, Ami, Amo, Amu;
		uint64_t Asa, Ase, Asi, Aso, Asu;
		uint64_t BCa, BCe, BCi, BCo, BCu;
		uint64_t Da, De, Di, Do, Du;
		uint64_t Eba, Ebe, Ebi, Ebo, Ebu;
		uint64_t Ega, Ege, Egi, Ego, Egu;
		uint64_t Eka, Eke, Eki, Eko, Eku;
		uint64_t Ema, Eme, Emi, Emo, Emu;
		uint64_t Esa, Ese, Esi, Eso, Esu;
#define    ROL ROTL64

		//copyFromState(A, state)
		Aba = state[0];
		Abe = state[1];
		Abi = state[2];
		Abo = state[3];
		Abu = state[4];
		Aga = state[5];
		Age = state[6];
		Agi = state[7];
		Ago = state[8];
		Agu = state[9];
		Aka = state[10];
		Ake = state[11];
		Aki = state[12];
		Ako = state[13];
		Aku = state[14];
		Ama = state[15];
		Ame = state[16];
		Ami = state[17];
		Amo = state[18];
		Amu = state[19];
		Asa = state[20];
		Ase = state[21];
		Asi = state[22];
		Aso = state[23];
		Asu = state[24];

		for (int round = 0; round < 24; round += 2)
		{
			//    prepareTheta
			BCa = Aba^Aga^Aka^Ama^Asa;
			BCe = Abe^Age^Ake^Ame^Ase;
			BCi = Abi^Agi^Aki^Ami^Asi;
			BCo = Abo^Ago^Ako^Amo^Aso;
			BCu = Abu^Agu^Aku^Amu^Asu;

			//thetaRhoPiChiIotaPrepareTheta(round  , A, E)
			Da = BCu^ROL(BCe, 1);
			De = BCa^ROL(BCi, 1);
			Di = BCe^ROL(BCo, 1);
			Do = BCi^ROL(BCu, 1);
			Du = BCo^ROL(BCa, 1);

			Aba ^= Da;
			BCa = Aba;
			Age ^= De;
			BCe = ROL(Age, 44);
			Aki ^= Di;
			BCi = ROL(Aki, 43);
			Amo ^= Do;
			BCo = ROL(Amo, 21);
			Asu ^= Du;
			BCu = ROL(Asu, 14);
			Eba = BCa ^ ((~BCe)&  BCi);
			Eba ^= keccak_round_constants[round];
			Ebe = BCe ^ ((~BCi)&  BCo);
			Ebi = BCi ^ ((~BCo)&  BCu);
			Ebo = BCo ^ ((~BCu)&  BCa);
			Ebu = BCu ^ ((~BCa)&  BCe);

			Abo ^= Do;
			BCa = ROL(Abo, 28);
			Agu ^= Du;
			BCe = ROL(Agu, 20);
			Aka ^= Da;
			BCi = ROL(Aka, 3);
			Ame ^= De;
			BCo = ROL(Ame, 45);
			Asi ^= Di;
			BCu = ROL(Asi, 61);
			Ega = BCa ^ ((~BCe)&  BCi);
			Ege = BCe ^ ((~BCi)&  BCo);
			Egi = BCi ^ ((~BCo)&  BCu);
			Ego = BCo ^ ((~BCu)&  BCa);
			Egu = BCu ^ ((~BCa)&  BCe);

			Abe ^= De;
			BCa = ROL(Abe, 1);
			Agi ^= Di;
			BCe = ROL(Agi, 6);
			Ako ^= Do;
			BCi = ROL(Ako, 25);
			Amu ^= Du;
			BCo = ROL(Amu, 8);
			Asa ^= Da;
			BCu = ROL(Asa, 18);
			Eka = BCa ^ ((~BCe)&  BCi);
			Eke = BCe ^ ((~BCi)&  BCo);
			Eki = BCi ^ ((~BCo)&  BCu);
			Eko = BCo ^ ((~BCu)&  BCa);
			Eku = BCu ^ ((~BCa)&  BCe);

			Abu ^= Du;
			BCa = ROL(Abu, 27);
			Aga ^= Da;
			BCe = ROL(Aga, 36);
			Ake ^= De;
			BCi = ROL(Ake, 10);
			Ami ^= Di;
			BCo = ROL(Ami, 15);
			Aso ^= Do;
			BCu = ROL(Aso, 56);
			Ema = BCa ^ ((~BCe)&  BCi);
			Eme = BCe ^ ((~BCi)&  BCo);
			Emi = BCi ^ ((~BCo)&  BCu);
			Emo = BCo ^ ((~BCu)&  BCa);
			Emu = BCu ^ ((~BCa)&  BCe);

			Abi ^= Di;
			BCa = ROL(Abi, 62);
			Ago ^= Do;
			BCe = ROL(Ago, 55);
			Aku ^= Du;
			BCi = ROL(Aku, 39);
			Ama ^= Da;
			BCo = ROL(Ama, 41);
			Ase ^= De;
			BCu = ROL(Ase, 2);
			Esa = BCa ^ ((~BCe)&  BCi);
			Ese = BCe ^ ((~BCi)&  BCo);
			Esi = BCi ^ ((~BCo)&  BCu);
			Eso = BCo ^ ((~BCu)&  BCa);
			Esu = BCu ^ ((~BCa)&  BCe);

			//    prepareTheta
			BCa = Eba^Ega^Eka^Ema^Esa;
			BCe = Ebe^Ege^Eke^Eme^Ese;
			BCi = Ebi^Egi^Eki^Emi^Esi;
			BCo = Ebo^Ego^Eko^Emo^Eso;
			BCu = Ebu^Egu^Eku^Emu^Esu;

			//thetaRhoPiChiIotaPrepareTheta(round+1, E, A)
			Da = BCu^ROL(BCe, 1);
			De = BCa^ROL(BCi, 1);
			Di = BCe^ROL(BCo, 1);
			Do = BCi^ROL(BCu, 1);
			Du = BCo^ROL(BCa, 1);

			Eba ^= Da;
			BCa = Eba;
			Ege ^= De;
			BCe = ROL(Ege, 44);
			Eki ^= Di;
			BCi = ROL(Eki, 43);
			Emo ^= Do;
			BCo = ROL(Emo, 21);
			Esu ^= Du;
			BCu = ROL(Esu, 14);
			Aba = BCa ^ ((~BCe)&  BCi);
			Aba ^= keccak_round_constants[round + 1];
			Abe = BCe ^ ((~BCi)&  BCo);
			Abi = BCi ^ ((~BCo)&  BCu);
			Abo = BCo ^ ((~BCu)&  BCa);
			Abu = BCu ^ ((~BCa)&  BCe);

			Ebo ^= Do;
			BCa = ROL(Ebo, 28);
			Egu ^= Du;
			BCe = ROL(Egu, 20);
			Eka ^= Da;
			BCi = ROL(Eka, 3);
			Eme ^= De;
			BCo = ROL(Eme, 45);
			Esi ^= Di;
			BCu = ROL(Esi, 61);
			Aga = BCa ^ ((~BCe)&  BCi);
			Age = BCe ^ ((~BCi)&  BCo);
			Agi = BCi ^ ((~BCo)&  BCu);
			Ago = BCo ^ ((~BCu)&  BCa);
			Agu = BCu ^ ((~BCa)&  BCe);

			Ebe ^= De;
			BCa = ROL(Ebe, 1);
			Egi ^= Di;
			BCe = ROL(Egi, 6);
			Eko ^= Do;
			BCi = ROL(Eko, 25);
			Emu ^= Du;
			BCo = ROL(Emu, 8);
			Esa ^= Da;
			BCu = ROL(Esa, 18);
			Aka = BCa ^ ((~BCe)&  BCi);
			Ake = BCe ^ ((~BCi)&  BCo);
			Aki = BCi ^ ((~BCo)&  BCu);
			Ako = BCo ^ ((~BCu)&  BCa);
			Aku = BCu ^ ((~BCa)&  BCe);

			Ebu ^= Du;
			BCa = ROL(Ebu, 27);
			Ega ^= Da;
			BCe = ROL(Ega, 36);
			Eke ^= De;
			BCi = ROL(Eke, 10);
			Emi ^= Di;
			BCo = ROL(Emi, 15);
			Eso ^= Do;
			BCu = ROL(Eso, 56);
			Ama = BCa ^ ((~BCe)&  BCi);
			Ame = BCe ^ ((~BCi)&  BCo);
			Ami = BCi ^ ((~BCo)&  BCu);
			Amo = BCo ^ ((~BCu)&  BCa);
			Amu = BCu ^ ((~BCa)&  BCe);

			Ebi ^= Di;
			BCa = ROL(Ebi, 62);
			Ego ^= Do;
			BCe = ROL(Ego, 55);
			Eku ^= Du;
			BCi = ROL(Eku, 39);
			Ema ^= Da;
			BCo = ROL(Ema, 41);
			Ese ^= De;
			BCu = ROL(Ese, 2);
			Asa = BCa ^ ((~BCe)&  BCi);
			Ase = BCe ^ ((~BCi)&  BCo);
			Asi = BCi ^ ((~BCo)&  BCu);
			Aso = BCo ^ ((~BCu)&  BCa);
			Asu = BCu ^ ((~BCa)&  BCe);
		}

		//copyToState(state, A)
		state[0] = Aba;
		state[1] = Abe;
		state[2] = Abi;
		state[3] = Abo;
		state[4] = Abu;
		state[5] = Aga;
		state[6] = Age;
		state[7] = Agi;
		state[8] = Ago;
		state[9] = Agu;
		state[10] = Aka;
		state[11] = Ake;
		state[12] = Aki;
		state[13] = Ako;
		state[14] = Aku;
		state[15] = Ama;
		state[16] = Ame;
		state[17] = Ami;
		state[18] = Amo;
		state[19] = Amu;
		state[20] = Asa;
		state[21] = Ase;
		state[22] = Asi;
		state[23] = Aso;
		state[24] = Asu;

#undef    ROL
	}
}



static __forceinline__ void keccak_block_host(uint64_t *s, const uint64_t *keccak_round_constants) {
    size_t i;
    uint64_t t[5], u[5], v, w;

    /* absorb input */    
    
    for (i = 0; i < 24; i++) {
        /* theta: c = a[0,i] ^ a[1,i] ^ .. a[4,i] */
        t[0] = s[0] ^ s[5] ^ s[10] ^ s[15] ^ s[20];
        t[1] = s[1] ^ s[6] ^ s[11] ^ s[16] ^ s[21];
        t[2] = s[2] ^ s[7] ^ s[12] ^ s[17] ^ s[22];
        t[3] = s[3] ^ s[8] ^ s[13] ^ s[18] ^ s[23];
        t[4] = s[4] ^ s[9] ^ s[14] ^ s[19] ^ s[24];

        /* theta: d[i] = c[i+4] ^ rotl(c[i+1],1) */
        u[0] = t[4] ^ ROTL64(t[1], 1);
        u[1] = t[0] ^ ROTL64(t[2], 1);
        u[2] = t[1] ^ ROTL64(t[3], 1);
        u[3] = t[2] ^ ROTL64(t[4], 1);
        u[4] = t[3] ^ ROTL64(t[0], 1);

        /* theta: a[0,i], a[1,i], .. a[4,i] ^= d[i] */
        s[0] ^= u[0]; s[5] ^= u[0]; s[10] ^= u[0]; s[15] ^= u[0]; s[20] ^= u[0];
        s[1] ^= u[1]; s[6] ^= u[1]; s[11] ^= u[1]; s[16] ^= u[1]; s[21] ^= u[1];
        s[2] ^= u[2]; s[7] ^= u[2]; s[12] ^= u[2]; s[17] ^= u[2]; s[22] ^= u[2];
        s[3] ^= u[3]; s[8] ^= u[3]; s[13] ^= u[3]; s[18] ^= u[3]; s[23] ^= u[3];
        s[4] ^= u[4]; s[9] ^= u[4]; s[14] ^= u[4]; s[19] ^= u[4]; s[24] ^= u[4];

        /* rho pi: b[..] = rotl(a[..], ..) */
        v = s[ 1];
        s[ 1] = ROTL64(s[ 6], 44);
        s[ 6] = ROTL64(s[ 9], 20);
        s[ 9] = ROTL64(s[22], 61);
        s[22] = ROTL64(s[14], 39);
        s[14] = ROTL64(s[20], 18);
        s[20] = ROTL64(s[ 2], 62);
        s[ 2] = ROTL64(s[12], 43);
        s[12] = ROTL64(s[13], 25);
        s[13] = ROTL64(s[19],  8);
        s[19] = ROTL64(s[23], 56);
        s[23] = ROTL64(s[15], 41);
        s[15] = ROTL64(s[ 4], 27);
        s[ 4] = ROTL64(s[24], 14);
        s[24] = ROTL64(s[21],  2);
        s[21] = ROTL64(s[ 8], 55);
        s[ 8] = ROTL64(s[16], 45);
        s[16] = ROTL64(s[ 5], 36);
        s[ 5] = ROTL64(s[ 3], 28);
        s[ 3] = ROTL64(s[18], 21);
        s[18] = ROTL64(s[17], 15);
        s[17] = ROTL64(s[11], 10);
        s[11] = ROTL64(s[ 7],  6);
        s[ 7] = ROTL64(s[10],  3);
        s[10] = ROTL64(    v,  1);

        /* chi: a[i,j] ^= ~b[i,j+1] & b[i,j+2] */
        v = s[ 0]; w = s[ 1]; s[ 0] ^= (~w) & s[ 2]; s[ 1] ^= (~s[ 2]) & s[ 3]; s[ 2] ^= (~s[ 3]) & s[ 4]; s[ 3] ^= (~s[ 4]) & v; s[ 4] ^= (~v) & w;
        v = s[ 5]; w = s[ 6]; s[ 5] ^= (~w) & s[ 7]; s[ 6] ^= (~s[ 7]) & s[ 8]; s[ 7] ^= (~s[ 8]) & s[ 9]; s[ 8] ^= (~s[ 9]) & v; s[ 9] ^= (~v) & w;
        v = s[10]; w = s[11]; s[10] ^= (~w) & s[12]; s[11] ^= (~s[12]) & s[13]; s[12] ^= (~s[13]) & s[14]; s[13] ^= (~s[14]) & v; s[14] ^= (~v) & w;
        v = s[15]; w = s[16]; s[15] ^= (~w) & s[17]; s[16] ^= (~s[17]) & s[18]; s[17] ^= (~s[18]) & s[19]; s[18] ^= (~s[19]) & v; s[19] ^= (~v) & w;
        v = s[20]; w = s[21]; s[20] ^= (~w) & s[22]; s[21] ^= (~s[22]) & s[23]; s[22] ^= (~s[23]) & s[24]; s[23] ^= (~s[24]) & v; s[24] ^= (~v) & w;

        /* iota: a[0,0] ^= round constant */
        s[0] ^= keccak_round_constants[i];
    }
}



 __constant__ uint64_t c_PaddedMessage80[16]; // padded message (80 bytes + padding)



__global__ __launch_bounds__(128,4) void  m7_keccak512_gpu_hash_120(int threads, uint32_t startNounce, uint64_t *outputHash)
{

    int thread = (blockDim.x * blockIdx.x + threadIdx.x);
    if (thread < threads)
    {
        
		uint32_t nounce = startNounce + thread;

         uint64_t state[25];

        #pragma unroll 16
		 for (int i=9;i<25;i++) {state[i]=stateo[i];}

		state[0] = xor1(stateo[0],c_PaddedMessage80[9]);
		state[1] = xor1(stateo[1],c_PaddedMessage80[10]);
		state[2] = xor1(stateo[2],c_PaddedMessage80[11]);
		state[3] = xor1(stateo[3],c_PaddedMessage80[12]);
		state[4] = xor1(stateo[4],c_PaddedMessage80[13]);
		state[5] = xor1(stateo[5],REPLACE_HIWORD(c_PaddedMessage80[14],nounce));
		state[6] = xor1(stateo[6],c_PaddedMessage80[15]);
		state[7] = stateo[7];
		state[8] = xor1(stateo[8],0x8000000000000000);
		 
//		keccak_block(state,RC);
		keccak_blockv3(state, RC);
#pragma unroll 8 
for (int i=0;i<8;i++) {outputHash[i*threads+thread]=state[i];}


	} //thread
}

__global__ __launch_bounds__(256,3) void ziftr_keccak512_gpu_hash_80(int threads, uint32_t startNounce, uint32_t *outputHash,uint32_t *test)
{

    int thread = (blockDim.x * blockIdx.x + threadIdx.x);
    if (thread < threads)
    {
        
		
        uint32_t nounce = startNounce +  thread ; // original implementation
	//	uint32_t nounce = cuda_swab32(nounce2);
       
		 uint2 ustate[25];
         #pragma unroll 25;
		 for(int i=0;i<25;i++) {ustate[i]=vectorize(stateo[i]);}
		 
		 uint2 addnonce; 
        LOHI(addnonce.x,addnonce.y,c_PaddedMessage80[9]);
        addnonce.y = nounce;
		ustate[0] ^= addnonce;
		ustate[1] ^= vectorize(c_PaddedMessage80[10]);
		ustate[8] ^= make_uint2(0x0,0x80000000);
		 
		keccak_blockv4(ustate, RC);

	 

#pragma unroll 8 
		for (int i = 0; i<8; i++) 
			 ((uint64_t*)(outputHash+16*thread))[i] = devectorize(ustate[i]);
		
		(test+thread)[0] = ((uint32_t*)arrOrder)[ustate[0].x % 24];
		

	} //thread
}

__global__ __launch_bounds__(256, 3) void ziftr_keccak512_gpu_hash_80_round2(int threads, uint32_t startNounce, uint32_t *outputHash, uint32_t *test)
{

	int thread = (blockDim.x * blockIdx.x + threadIdx.x);
	if (thread < threads)
	{


		uint32_t nounce = startNounce + thread; 

		uint2 ustate[25];

		#pragma unroll
		for (int i = 9; i<25; i++) { ustate[i] = make_uint2(0,0); }
        #pragma unroll
		for (int i = 0; i<9; i++) {  ustate[i] = vectorize(c_PaddedMessage80[i]); }
		ustate[0].x |= (0xFFFF0000 & ((outputHash + 16 * thread)[0] & 0xFFFF0000));
		
		keccak_blockv4(ustate, RC);

		uint2 addnonce;
		LOHI(addnonce.x, addnonce.y, c_PaddedMessage80[9]);
		addnonce.y = nounce;
		ustate[0] ^= addnonce;
		ustate[1] ^= vectorize(c_PaddedMessage80[10]);
		ustate[8] ^= make_uint2(0x0, 0x80000000);

		keccak_blockv4(ustate, RC);

        #pragma unroll 8 
		for (int i = 0; i<8; i++)
			((uint64_t*)(outputHash + 16 * thread))[i] = devectorize(ustate[i]);

		(test + thread)[0] = ((uint32_t*)arrOrder)[ustate[0].x % 24];
	} //thread
}

__global__ __launch_bounds__(128, 2) void ziftr_keccak512_gpu_hash_80_v30(int threads, uint32_t startNounce, uint32_t *outputHash, uint32_t *test)
{

	int thread = (blockDim.x * blockIdx.x + threadIdx.x);
	if (thread < threads)
	{


		uint32_t nounce = startNounce + thread; // original implementation
		//	uint32_t nounce = cuda_swab32(nounce2);

		uint64_t ustate[25];
#pragma unroll 25;
		for (int i = 0; i<25; i++) { ustate[i] = stateo[i]; }

		uint2 addnonce;
		LOHI(addnonce.x, addnonce.y, c_PaddedMessage80[9]);
		addnonce.y = nounce;
		ustate[0] ^= devectorize(addnonce);
		ustate[1] ^= c_PaddedMessage80[10];
		ustate[8] ^= 0x8000000000000000;

		keccak_block(ustate, RC);



#pragma unroll 8 
		for (int i = 0; i<8; i++)
			((uint64_t*)(outputHash + 16 * thread))[i] = ustate[i];

		(test + thread)[0] = ((uint32_t*)arrOrder)[((uint32_t*)ustate)[0] % 24];


	} //thread
}

__global__ __launch_bounds__(128, 2) void ziftr_keccak512_gpu_hash_80_round2_v30(int threads, uint32_t startNounce, uint32_t *outputHash, uint32_t *test)
{

	int thread = (blockDim.x * blockIdx.x + threadIdx.x);
	if (thread < threads)
	{


		uint32_t nounce = startNounce + thread;

		uint64_t ustate[25];

#pragma unroll
		for (int i = 9; i<25; i++) { ustate[i] = 0; }
#pragma unroll
		for (int i = 0; i<9; i++) { ustate[i] = c_PaddedMessage80[i]; }
		((uint32_t*)ustate)[0] |= (0xFFFF0000 & ((outputHash + 16 * thread)[0] & 0xFFFF0000));

		keccak_block(ustate, RC);

		uint2 addnonce;
		LOHI(addnonce.x, addnonce.y, c_PaddedMessage80[9]);
		addnonce.y = nounce;
		ustate[0] ^= devectorize(addnonce);
		ustate[1] ^= c_PaddedMessage80[10];
		ustate[8] ^=  0x8000000000000000;

		keccak_block(ustate, RC);

#pragma unroll 8 
		for (int i = 0; i<8; i++)
			((uint64_t*)(outputHash + 16 * thread))[i] = ustate[i];

		(test + thread)[0] = ((uint32_t*)arrOrder)[((uint32_t*)ustate)[0] % 24];
	} //thread
}


void m7_keccak512_cpu_init(int thr_id, int threads)
{
    	
	hipMemcpyToSymbol(HIP_SYMBOL( RC),cpu_RC,sizeof(cpu_RC),0,hipMemcpyHostToDevice);	
} 

__host__ void m7_keccak512_setBlock_120(void *pdata)
{

	unsigned char PaddedMessage[128];
	uint8_t ending =0x01;
	memcpy(PaddedMessage, pdata, 122);
	memset(PaddedMessage+122,ending,1); 
	memset(PaddedMessage+123, 0, 5); 
	hipMemcpyToSymbol(HIP_SYMBOL( c_PaddedMessage80), PaddedMessage, 16*sizeof(uint64_t), 0, hipMemcpyHostToDevice);
	uint64_t* alt_data = (uint64_t*) pdata;
         uint64_t state[25];
		 for(int i=0;i<25;i++) {state[i]=0;}
           alt_data[0] &= (~0xFFFF0000);    //// attention modif for ziftrcoin

		for (int i=0;i<9;i++) {state[i]  ^= alt_data[i];}
		
		keccak_block_host(state,cpu_RC);

		hipMemcpyToSymbol(HIP_SYMBOL(stateo), state, 25*sizeof(uint64_t), 0, hipMemcpyHostToDevice);

}


__host__ void m7_keccak512_cpu_hash(int thr_id, int threads, uint32_t startNounce, uint64_t *d_hash, int order)
{
    const int threadsperblock = 128;

    dim3 grid(threads/threadsperblock);
    dim3 block(threadsperblock);

    size_t shared_size = 0;

    m7_keccak512_gpu_hash_120<<<grid, block, shared_size>>>(threads, startNounce, d_hash);
    MyStreamSynchronize(NULL, order, thr_id);
}

__host__ void m7_keccak512_setBlock_80(void *pdata)
{

	unsigned char PaddedMessage[128];
	uint8_t ending =0x01;
	memcpy(PaddedMessage, pdata, 80);
	memset(PaddedMessage+80,ending,1); 
	memset(PaddedMessage+81, 0, 47); 
	hipMemcpyToSymbol(HIP_SYMBOL(c_PaddedMessage80), PaddedMessage, 16*sizeof(uint64_t), 0, hipMemcpyHostToDevice);
	uint64_t* alt_data = (uint64_t*) pdata;
         uint64_t state[25];
		 for(int i=0;i<25;i++) {state[i]=0;}
		for (int i=0;i<9;i++) {state[i]  ^= alt_data[i];}
		keccak_block_host(state,cpu_RC);

		hipMemcpyToSymbol(HIP_SYMBOL(stateo), state, 25*sizeof(uint64_t), 0, hipMemcpyHostToDevice);

}

__host__ void ziftr_keccak512_setBlock_80(void *pdata)
{

	unsigned char PaddedMessage[128];
	uint8_t ending = 0x01;
	memcpy(PaddedMessage, pdata, 80);
	memset(PaddedMessage + 80, ending, 1);
	memset(PaddedMessage + 81, 0, 47);
	hipMemcpyToSymbol(HIP_SYMBOL(c_PaddedMessage80), PaddedMessage, 16 * sizeof(uint64_t), 0, hipMemcpyHostToDevice);
	uint64_t* alt_data = (uint64_t*)pdata;
	uint64_t state[25];
	for (int i = 0; i<25; i++) { state[i] = 0; }
	state[0] = alt_data[0] & (~0xffff0000);
	for (int i = 1; i<9; i++) { state[i] = alt_data[i]; }
	keccak_block_host(state, cpu_RC);

	hipMemcpyToSymbol(HIP_SYMBOL(stateo), state, 25 * sizeof(uint64_t), 0, hipMemcpyHostToDevice);

}


__host__ void ziftr_keccak512_cpu_hash_80(int thr_id, int threads, uint32_t startNounce, uint32_t *d_hash,uint32_t* d_test, int order)
{
    const int threadsperblock = 256;

	dim3 grid((threads + threadsperblock - 1) / threadsperblock);
    dim3 block(threadsperblock);

	const int threadsperblock2 = 128;
	dim3 grid2((threads + threadsperblock2 - 1) / threadsperblock2);
	dim3 block2(threadsperblock2);


    size_t shared_size = 0;
	if (compute_version[thr_id]>30) 
    ziftr_keccak512_gpu_hash_80<<<grid, block, shared_size>>>(threads, startNounce, d_hash, d_test);
    else 
	ziftr_keccak512_gpu_hash_80_v30 << <grid2, block2, shared_size >> >(threads, startNounce, d_hash, d_test);
    MyStreamSynchronize(NULL, order, thr_id);
}

__host__ void ziftr_keccak512_cpu_hash_80_round2(int thr_id, int threads, uint32_t startNounce, uint32_t *d_hash, uint32_t* d_test, int order)
{
	const int threadsperblock = 256;
	dim3 grid((threads + threadsperblock - 1) / threadsperblock);
	dim3 block(threadsperblock);

	const int threadsperblock2 = 128;
	dim3 grid2((threads + threadsperblock2 - 1) / threadsperblock2);
	dim3 block2(threadsperblock2);


	size_t shared_size = 0;
	if (compute_version[thr_id]>30)
	ziftr_keccak512_gpu_hash_80_round2 << <grid, block, shared_size >> >(threads, startNounce, d_hash, d_test);
    else
		ziftr_keccak512_gpu_hash_80_round2_v30 << <grid2, block2, shared_size >> >(threads, startNounce, d_hash, d_test);
 
	MyStreamSynchronize(NULL, order, thr_id);
}
