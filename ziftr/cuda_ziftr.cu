#include "hip/hip_runtime.h"

extern "C"
{
#include "sph/sph_blake.h"
#include "sph/sph_groestl.h"
#include "sph/sph_skein.h"
#include "sph/sph_jh.h"
#include "sph/sph_keccak.h"
#include "miner.h"
}

#include <stdint.h>

// aus cpu-miner.c
extern int device_map[8];

// Speicher f�r Input/Output der verketteten Hashfunktionen
static uint32_t *d_hash[8];
static uint32_t *d_test[8];

extern void quark_blake512_cpu_init(int thr_id, int threads);
extern void quark_blake512_cpu_setBlock_80(void *pdata);

extern void ziftr_blake512_cpu_hash_64(int thr_id, int threads, uint32_t startNounce, uint32_t *d_outputHash, uint32_t *d_test, uint32_t table, int order);
extern void quark_groestl512_cpu_init(int thr_id, int threads);
extern void quark_groestl512_sm20_init(int thr_id, uint32_t threads);
extern void ziftr_groestl512_cpu_hash_64(int thr_id, int threads, uint32_t startNounce, uint32_t *d_hash, uint32_t *d_test, uint32_t table, int order);
extern void ziftr_groestl512_sm20_hash_64(int thr_id, uint32_t threads, uint32_t startNounce, uint32_t *d_hash, uint32_t *d_test, uint32_t table, int order);
extern void quark_jh512_cpu_init(int thr_id, int threads);
extern void ziftr_jh512_cpu_hash_64(int thr_id, int threads, uint32_t startNounce, uint32_t *d_hash, uint32_t *d_test, uint32_t table, int order);

extern void quark_skein512_cpu_init(int thr_id, int threads);
extern void ziftr_skein512_cpu_hash_64(int thr_id, int threads, uint32_t startNounce, uint32_t *d_hash, uint32_t *d_test, uint32_t table, int order);

extern void m7_keccak512_setBlock_80(void *pdata);
extern void ziftr_keccak512_setBlock_80(void *pdata);
extern void ziftr_keccak512_cpu_hash_80(int thr_id, int threads, uint32_t startNounce, uint32_t *d_hash, uint32_t *d_test,int order);
extern void ziftr_keccak512_cpu_hash_80_round2(int thr_id, int threads, uint32_t startNounce, uint32_t *d_hash, uint32_t *d_test, int order);

extern void m7_keccak512_cpu_init(int thr_id, int threads);


extern void quark_check_cpu_init(int thr_id, int threads);
extern void quark_check_cpu_setTarget(const void *ptarget);
extern uint32_t quark_check_cpu_hash_64(int thr_id, int threads, uint32_t startNounce, uint32_t *d_nonceVector, uint32_t *d_inputHash, int order);

inline void zr5hash(void *state, const void *input)
{
    sph_blake512_context ctx_blake;
    sph_groestl512_context ctx_groestl;
    sph_jh512_context ctx_jh;
    sph_keccak512_context ctx_keccak;
    sph_skein512_context ctx_skein;
	uint32_t hash[16];
	static const int BLAKE = 0;
	static const int GROESTL = 1;
	static const int JH = 2;
	static const int SKEIN = 3;
	static const int arrOrder[][4] =
	{
		{ 0, 1, 2, 3 },
		{ 0, 1, 3, 2 },
		{ 0, 2, 1, 3 },
		{ 0, 2, 3, 1 },
		{ 0, 3, 1, 2 },
		{ 0, 3, 2, 1 },
		{ 1, 0, 2, 3 },
		{ 1, 0, 3, 2 },
		{ 1, 2, 0, 3 },
		{ 1, 2, 3, 0 },
		{ 1, 3, 0, 2 },
		{ 1, 3, 2, 0 },
		{ 2, 0, 1, 3 },
		{ 2, 0, 3, 1 },
		{ 2, 1, 0, 3 },
		{ 2, 1, 3, 0 },
		{ 2, 3, 0, 1 },
		{ 2, 3, 1, 0 },
		{ 3, 0, 1, 2 },
		{ 3, 0, 2, 1 },
		{ 3, 1, 0, 2 },
		{ 3, 1, 2, 0 },
		{ 3, 2, 0, 1 },
		{ 3, 2, 1, 0 }
	};

	sph_keccak512_init(&ctx_keccak);
	sph_keccak512(&ctx_keccak, input, 80);
	sph_keccak512_close(&ctx_keccak, hash);
	uint32_t nOrder = hash[0] % (sizeof(arrOrder)/sizeof((arrOrder)[0]));
	int nSize = 64;
	
	for (unsigned int i = 0; i < 4; i++)
	{



		switch (arrOrder[nOrder][i])
		{
		case BLAKE:
			sph_blake512_init(&ctx_blake);
			sph_blake512(&ctx_blake, hash, nSize);
			sph_blake512_close(&ctx_blake, hash);

			break;
		case GROESTL:
			sph_groestl512_init(&ctx_groestl);
			sph_groestl512(&ctx_groestl, hash, nSize);
			sph_groestl512_close(&ctx_groestl, hash);
			
			break;
		case JH:
			sph_jh512_init(&ctx_jh);
			sph_jh512(&ctx_jh, hash, nSize);
			sph_jh512_close(&ctx_jh, hash);
			
			break;
		case SKEIN:
			sph_skein512_init(&ctx_skein);
			sph_skein512(&ctx_skein, hash, nSize);
			sph_skein512_close(&ctx_skein, hash);
			
			break;
		default:
			break;
		}
	}
    memcpy(state, hash, 64);
}


extern bool opt_benchmark;
extern int compute_version[8];

extern "C" int scanhash_zr5(int thr_id, uint32_t *pdata,
   const  uint32_t *ptarget, uint32_t max_nonce,
    unsigned long *hashes_done)
{
	const uint32_t first_nonce = pdata[19];

	if (opt_benchmark)
		((uint32_t*)ptarget)[7] = 0x0000ff;

	const uint32_t Htarg = ptarget[7];


	const int throughput = 256 * 4096 * 2;

	static bool init[8] = {0,0,0,0,0,0,0,0};
	if (!init[thr_id])
	{
		hipSetDevice(device_map[thr_id]);

		
		hipMalloc(&d_hash[thr_id], 16 * sizeof(uint32_t) * throughput);
		hipMalloc(&d_test[thr_id],      sizeof(uint32_t) * throughput);

		quark_blake512_cpu_init(thr_id, throughput);
		if (compute_version[thr_id]>30)
		quark_groestl512_cpu_init(thr_id, throughput);
        else
		quark_groestl512_sm20_init(thr_id,throughput);

		quark_jh512_cpu_init(thr_id, throughput);
		m7_keccak512_cpu_init(thr_id, throughput);
		quark_skein512_cpu_init(thr_id, throughput);
		quark_check_cpu_init(thr_id, throughput);
		init[thr_id] = true;
	}
 
	uint32_t endiandata[20],safedata[20];
	for (int k = 0; k < 20; k++) {
		endiandata[k] = pdata[k];}


 m7_keccak512_setBlock_80((void*)endiandata);
	quark_check_cpu_setTarget(ptarget);



	do {
		int order = 0;
/// round one 
		ziftr_keccak512_cpu_hash_80(thr_id, throughput, pdata[19],d_hash[thr_id],d_test[thr_id], order++);

		ziftr_blake512_cpu_hash_64(thr_id, throughput, pdata[19],    d_hash[thr_id], d_test[thr_id], 0x00040000, order++); //0
		if (compute_version[thr_id]>30)
		ziftr_groestl512_cpu_hash_64(thr_id, throughput, pdata[19],  d_hash[thr_id], d_test[thr_id], 0x00010000, order++); //1
		else
		ziftr_groestl512_sm20_hash_64(thr_id, throughput, pdata[19], d_hash[thr_id], d_test[thr_id], 0x00010000, order++); //1

		ziftr_jh512_cpu_hash_64(thr_id, throughput, pdata[19],       d_hash[thr_id], d_test[thr_id], 0x00020000, order++);  //2
		ziftr_skein512_cpu_hash_64(thr_id, throughput, pdata[19],    d_hash[thr_id], d_test[thr_id], 0x00030000, order++);  //3

		ziftr_blake512_cpu_hash_64(thr_id, throughput, pdata[19],    d_hash[thr_id], d_test[thr_id], 0x00040001, order++); //0
		if (compute_version[thr_id]>30)
		ziftr_groestl512_cpu_hash_64(thr_id, throughput, pdata[19],  d_hash[thr_id], d_test[thr_id], 0x00010001, order++); //1
		else
		ziftr_groestl512_sm20_hash_64(thr_id, throughput, pdata[19], d_hash[thr_id], d_test[thr_id], 0x00010001, order++); //1

		ziftr_jh512_cpu_hash_64(thr_id, throughput, pdata[19],       d_hash[thr_id], d_test[thr_id], 0x00020001, order++);  //2
		ziftr_skein512_cpu_hash_64(thr_id, throughput, pdata[19],    d_hash[thr_id], d_test[thr_id], 0x00030001, order++);  //3

		ziftr_blake512_cpu_hash_64(thr_id, throughput, pdata[19],    d_hash[thr_id], d_test[thr_id], 0x00040002, order++); //0
		if (compute_version[thr_id]>30)
		ziftr_groestl512_cpu_hash_64(thr_id, throughput, pdata[19],  d_hash[thr_id], d_test[thr_id], 0x00010002, order++); //1
		else
		ziftr_groestl512_sm20_hash_64(thr_id, throughput, pdata[19], d_hash[thr_id], d_test[thr_id], 0x00010002, order++); //1

		ziftr_jh512_cpu_hash_64(thr_id, throughput, pdata[19],       d_hash[thr_id], d_test[thr_id], 0x00020002, order++);  //2
		ziftr_skein512_cpu_hash_64(thr_id, throughput, pdata[19],    d_hash[thr_id], d_test[thr_id], 0x00030002, order++);  //3

		ziftr_blake512_cpu_hash_64(thr_id, throughput, pdata[19],    d_hash[thr_id], d_test[thr_id], 0x00040003, order++); //0
		if (compute_version[thr_id]>30)
		ziftr_groestl512_cpu_hash_64(thr_id, throughput, pdata[19],  d_hash[thr_id], d_test[thr_id], 0x00010003, order++); //1
		else
		ziftr_groestl512_sm20_hash_64(thr_id, throughput, pdata[19], d_hash[thr_id], d_test[thr_id], 0x00010003, order++); //1

		ziftr_jh512_cpu_hash_64(thr_id, throughput, pdata[19],       d_hash[thr_id], d_test[thr_id], 0x00020003, order++);  //2
		ziftr_skein512_cpu_hash_64(thr_id, throughput, pdata[19],    d_hash[thr_id], d_test[thr_id], 0x00030003, order++);  //3


		/// round two 
		ziftr_keccak512_cpu_hash_80_round2(thr_id, throughput, pdata[19], d_hash[thr_id], d_test[thr_id], order++);

		ziftr_blake512_cpu_hash_64(thr_id, throughput, pdata[19], d_hash[thr_id], d_test[thr_id], 0x00040000, order++); //0
		if (compute_version[thr_id]>30)
		ziftr_groestl512_cpu_hash_64(thr_id, throughput, pdata[19], d_hash[thr_id], d_test[thr_id], 0x00010000, order++); //1
		else
		ziftr_groestl512_sm20_hash_64(thr_id, throughput, pdata[19], d_hash[thr_id], d_test[thr_id], 0x00010000, order++); //1

		ziftr_jh512_cpu_hash_64(thr_id, throughput, pdata[19], d_hash[thr_id], d_test[thr_id], 0x00020000, order++);  //2
		ziftr_skein512_cpu_hash_64(thr_id, throughput, pdata[19], d_hash[thr_id], d_test[thr_id], 0x00030000, order++);  //3

		ziftr_blake512_cpu_hash_64(thr_id, throughput, pdata[19], d_hash[thr_id], d_test[thr_id], 0x00040001, order++); //0
		if (compute_version[thr_id]>30)
		ziftr_groestl512_cpu_hash_64(thr_id, throughput, pdata[19], d_hash[thr_id], d_test[thr_id], 0x00010001, order++); //1
		else
		ziftr_groestl512_sm20_hash_64(thr_id, throughput, pdata[19], d_hash[thr_id], d_test[thr_id], 0x00010001, order++); //1

		ziftr_jh512_cpu_hash_64(thr_id, throughput, pdata[19], d_hash[thr_id], d_test[thr_id], 0x00020001, order++);  //2
		ziftr_skein512_cpu_hash_64(thr_id, throughput, pdata[19], d_hash[thr_id], d_test[thr_id], 0x00030001, order++);  //3

		ziftr_blake512_cpu_hash_64(thr_id, throughput, pdata[19], d_hash[thr_id], d_test[thr_id], 0x00040002, order++); //0
		if (compute_version[thr_id]>30)
		ziftr_groestl512_cpu_hash_64(thr_id, throughput, pdata[19], d_hash[thr_id], d_test[thr_id], 0x00010002, order++); //1
		else
		ziftr_groestl512_sm20_hash_64(thr_id, throughput, pdata[19], d_hash[thr_id], d_test[thr_id], 0x00010002, order++); //1

		ziftr_jh512_cpu_hash_64(thr_id, throughput, pdata[19], d_hash[thr_id], d_test[thr_id], 0x00020002, order++);  //2
		ziftr_skein512_cpu_hash_64(thr_id, throughput, pdata[19], d_hash[thr_id], d_test[thr_id], 0x00030002, order++);  //3

		ziftr_blake512_cpu_hash_64(thr_id, throughput, pdata[19], d_hash[thr_id], d_test[thr_id], 0x00040003, order++); //0
		if (compute_version[thr_id]>30)
		ziftr_groestl512_cpu_hash_64(thr_id, throughput, pdata[19], d_hash[thr_id], d_test[thr_id], 0x00010003, order++); //1
        else 
		ziftr_groestl512_sm20_hash_64(thr_id, throughput, pdata[19], d_hash[thr_id], d_test[thr_id], 0x00010003, order++); //1

		ziftr_jh512_cpu_hash_64(thr_id, throughput, pdata[19], d_hash[thr_id], d_test[thr_id], 0x00020003, order++);  //2
		ziftr_skein512_cpu_hash_64(thr_id, throughput, pdata[19], d_hash[thr_id], d_test[thr_id], 0x00030003, order++);  //3



 
		uint32_t foundNonce = quark_check_cpu_hash_64(thr_id, throughput, pdata[19], NULL, d_hash[thr_id], order++);
// 		foundNonce = 10+pdata[19];
		if  (foundNonce != 0xffffffff)
		{
			uint32_t hash1[16];
			endiandata[0] = pdata[0] & (~0xFFFF0000);
			endiandata[19] = foundNonce;
			zr5hash(hash1, endiandata);
			endiandata[0] = endiandata[0] | (0xFFFF0000 & (hash1[0] & 0xFFFF0000));
			zr5hash(hash1, endiandata);
			
			if (fulltest(hash1, ptarget)) {
				pdata[19] = foundNonce;
				pdata[0] = endiandata[0]; // need to export both nonce and pok value
				*hashes_done = foundNonce - first_nonce + 1;
				return 1;
             }
		}

		if (((uint64_t)pdata[19] + (uint64_t)throughput) > (uint64_t)UINT32_MAX) {
			pdata[19]=max_nonce;
		} else {
		pdata[19] += throughput;
        }

	} while (pdata[19] < max_nonce && !work_restart[thr_id].restart);
//	pdata[0] = pdata[0] & (~0xFFFF0000); // reset pok
	*hashes_done = pdata[19] - first_nonce + 1;
	return 0;
}
